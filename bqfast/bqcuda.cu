#include "hip/hip_runtime.h"
#undef _GLIBCXX_USE_INT128
#undef _GLIBCXX_ATOMIC_BUILTINS

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <string.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <thrust/adjacent_difference.h>
#include <thrust/extrema.h>

#include "bqcuda.h"

#define cudaSafeCall(call){   \
  hipError_t err = call;       \
  if(hipSuccess != err){     \
    fprintf(stderr, "%s(%i) : %s.\n", __FILE__, __LINE__, hipGetErrorString(err));   \
    exit(EXIT_FAILURE);       \
}}

#define MAX_THREADS_PER_BLOCK 1024
#define SCORE_THREADS_PER_BLOCK 256
#define TALLY_THREADS_PER_BLOCK 768
#define MAX_BLOCKS_PER_GRID 65535

//[C2.AllQVsModel]
#define Match              0.2627555
#define Mismatch          -1.09688872
#define MismatchS         -0.01637988
#define Branch            -0.60275947
#define BranchS           -0.02682689
#define DeletionN         -1.00012494
#define DeletionWithTag    0.06000148
#define DeletionWithTagS  -0.02579358
#define Nce               -0.15864559
#define NceS              -0.04403654
#define Merge             -1.02398814
#define MergeS            -0.12135255

//metrics
#define InsertionIdx 0
#define MergeIdx 1
#define DeletionIdx 2
#define DeletionTagIdx 3
#define SubstitutionIdx 4

//https://github.com/thrust/thrust/blob/master/examples/strided_range.cu
template <typename Iterator>
class strided_range
{
    public:

    typedef typename thrust::iterator_difference<Iterator>::type difference_type;

    struct stride_functor : public thrust::unary_function<difference_type,difference_type>
    {
        difference_type stride;

        stride_functor(difference_type stride)
            : stride(stride) {}

        __host__ __device__
        difference_type operator()(const difference_type& i) const
        {
            return stride * i;
        }
    };

    typedef typename thrust::counting_iterator<difference_type>                   CountingIterator;
    typedef typename thrust::transform_iterator<stride_functor, CountingIterator> TransformIterator;
    typedef typename thrust::permutation_iterator<Iterator,TransformIterator>     PermutationIterator;

    // type of the strided_range iterator
    typedef PermutationIterator iterator;

    // construct strided_range for the range [first,last)
    strided_range(Iterator first, Iterator last, difference_type stride)
        : first(first), last(last), stride(stride) {}

    iterator begin(void) const
    {
        return PermutationIterator(first, TransformIterator(CountingIterator(0), stride_functor(stride)));
    }

    iterator end(void) const
    {
        return begin() + ((last - first) + (stride - 1)) / stride;
    }

    protected:
    Iterator first;
    Iterator last;
    difference_type stride;
};

__host__ __device__ void indexToMutation(uint8_t *origTmplSeq, int tmplLen, uint8_t *mutated, int index) {

    int baseOrd[4] = {'A', 'C', 'G', 'T'};
    int tmpPos = index / 8;
    int mut = index % 8;
    int currentBasePos;

    for (int i = 0; i < tmpPos; i++) {
        mutated[i] = origTmplSeq[i];
    }

    for (int i = 0; i < 4; i++) {
        if (origTmplSeq[tmpPos] == baseOrd[i]) {
            currentBasePos = i;
        }
    }

    if (mut < 4) {
        mutated[tmpPos] = baseOrd[mut % 4];
        for (int i = tmpPos; i < tmplLen - 1; i++) {
            mutated[i + 1] = origTmplSeq[i];
        }
        mutated[tmplLen - 1] = '\0';
    } else if (mut >= 4 and mut < 7) {
        mutated[tmpPos] = baseOrd[(currentBasePos + ((mut-4+1) % 4)) % 4];
        for (int i = tmpPos + 1; i < tmplLen; i++) {
            mutated[i] = origTmplSeq[i];
        }
    } else {
        for (int i = tmpPos + 1; i < tmplLen; i++) {
            mutated[i-1] = origTmplSeq[i];
        }
        mutated[tmplLen - 1] = '\0';
    }

}

__device__ uint8_t ReadBase(uint8_t *readSeqs, size_t rs_pitch, int read_id, int read_pos) {
    return *((uint8_t*)((char*) readSeqs + read_pos * rs_pitch) + read_id);
}

__device__ uint8_t QVInfoVal(hipPitchedPtr qvInfo, int readLen, int metric, int read_id, int read_pos) {
    size_t pitch = qvInfo.pitch;
    size_t slicePitch = pitch * readLen;
    return *((uint8_t*)(((char*) qvInfo.ptr + metric * slicePitch) + read_pos * pitch) + read_id);
}

__global__ void ScoreMutations(uint8_t *origTmplSeq, uint8_t *readSeqs, size_t rs_pitch, hipPitchedPtr qvInfo, int tmplLen, int readLen, int numReads, double *results) {

    int block_global_index = SCORE_THREADS_PER_BLOCK * (blockIdx.y * gridDim.x + blockIdx.x);
    int thread_id = (blockDim.x * threadIdx.y) + threadIdx.x;
    int global_index = block_global_index + thread_id;
    if (global_index >= 8 * tmplLen * numReads) return;

    int mut_id = global_index / numReads;
    int read_id = global_index % numReads;

    uint8_t tmplSeq[128];

    indexToMutation(origTmplSeq, tmplLen, tmplSeq, mut_id);

    double alpha[4][130];

    double score = -FLT_MAX;
    int lastAlphaI, lastAlphaJ;
    double moveScore = 0;

    for (int i = 0; i < readLen + 1; i++) {

        int alphamodi = i % 4;
        int alphamodim1 = (i - 1) % 4;
        if (i > 0 and ReadBase(readSeqs, rs_pitch, read_id, i - 1) == 0) {
            lastAlphaI = alphamodim1;
            break;
        }

        for (int j = 0; j < tmplLen + 1; j++) {

            if (j > 0 and tmplSeq[j - 1] == 0) {
                lastAlphaJ = j - 1;
                break;
            }

            score = (-FLT_MAX);

            if (i == 0 and j == 0) {
                score = 0;
            }

            moveScore = -1;

            //Incorporate
            if (i > 0 and j > 0) {

                if (ReadBase(readSeqs, rs_pitch, read_id, i - 1) == tmplSeq[j - 1]) {
                    moveScore = alpha[alphamodim1][j - 1] + Match;
                } else {
                    moveScore = alpha[alphamodim1][j - 1] + Mismatch + MismatchS * QVInfoVal(qvInfo, readLen, SubstitutionIdx, read_id,  i - 1);
                }
                score = (moveScore > score) ? moveScore : score;
            }
            //Extra
            if (i > 0) {
                if (j < tmplLen and ReadBase(readSeqs, rs_pitch, read_id, i - 1) == tmplSeq[j]) {
                    moveScore = alpha[alphamodim1][j] + Branch + BranchS * QVInfoVal(qvInfo, readLen, InsertionIdx, read_id,  i - 1);
                } else {
                    moveScore = alpha[alphamodim1][j] + Nce + NceS * QVInfoVal(qvInfo, readLen, InsertionIdx, read_id,  i - 1);
                }
                score = (moveScore > score) ? moveScore : score;
            }

            //Delete
            if (j > 0) {
                if (i < readLen and QVInfoVal(qvInfo, readLen, DeletionTagIdx, read_id,  i) == tmplSeq[j - 1]) {
                    moveScore = alpha[alphamodi][j - 1] + DeletionWithTag + DeletionWithTagS * QVInfoVal(qvInfo, readLen, DeletionIdx, read_id,  i);
                } else {
                    moveScore = alpha[alphamodi][j - 1] + DeletionN;
                }
                score = (moveScore > score) ? moveScore : score;
            }

            //Merge
            if (i > 0 and j > 1) {
                if (! (ReadBase(readSeqs, rs_pitch, read_id, i - 1) == tmplSeq[j - 2] && ReadBase(readSeqs, rs_pitch, read_id, i - 1) == tmplSeq[j - 1])) {
                    //moveScore = alpha[i - 1][j - 2] + (-FLT_MAX);
                    moveScore = -FLT_MAX;
                } else {
                    moveScore = alpha[alphamodim1][j - 2] + Merge + MergeS * QVInfoVal(qvInfo, readLen, MergeIdx, read_id,  i - 1);
                }
                score = (moveScore > score) ? moveScore : score;
            }
            alpha[alphamodi][j] = score;
        }
    }

    results[global_index] = alpha[lastAlphaI][lastAlphaJ];

}



int run_bqcuda(uint8_t *origTmplSeq, uint8_t *polishedTmplSeq, uint8_t *readSeqs, uint8_t *qvInfo, double *results, double origTmplScore, int tmplLen, int readLen, int numReads, int numMetrics) {
    
    uint8_t *d_currentTmplSeq;
    uint8_t *d_readSeqs;
    hipPitchedPtr d_qvInfo;
    
    size_t rs_pitch;

    int template_mutations = 8 * tmplLen;
    double currentTmplScore = origTmplScore;
    
    /** Copy data to GPU **/

    // Copy original template
    memcpy(polishedTmplSeq, origTmplSeq, tmplLen * sizeof(uint8_t));
    cudaSafeCall( hipMalloc(&d_currentTmplSeq, tmplLen * sizeof(uint8_t)) );
    
    // Copy read seqs
    cudaSafeCall( hipMallocPitch(&d_readSeqs, &rs_pitch, numReads * sizeof(uint8_t), readLen * sizeof(uint8_t)) );
    
    for (unsigned int i = 0; i < readLen; i++) {
      cudaSafeCall( hipMemcpy((uint8_t*)((char*) d_readSeqs + i * rs_pitch), readSeqs + i * numReads, sizeof(uint8_t) * numReads, hipMemcpyHostToDevice) );
    }

    // Copy QV info
    cudaSafeCall( hipMalloc3D(&d_qvInfo, make_hipExtent(numReads * sizeof(uint8_t), readLen, numMetrics)) );
    
    size_t pitch = d_qvInfo.pitch;
    size_t slicePitch = pitch * readLen;
    
    for (int z = 0; z < numMetrics; z++) {
        char* slice = (char*) d_qvInfo.ptr + z * slicePitch;
        for (int y = 0; y < readLen; y++) {
            uint8_t* row = (uint8_t*)(slice + y * pitch);
            cudaSafeCall( hipMemcpy(row, qvInfo + z * readLen * numReads + y * numReads, sizeof(uint8_t) * numReads, hipMemcpyHostToDevice) );
        }
    }

    /** Allocate working space on GPU **/

    // Create storage for P(read | mutated template) on GPU
    double *d_results;
    cudaSafeCall( hipMalloc(&d_results, template_mutations * numReads * sizeof(double)) );
    thrust::device_ptr<double> d_results_start(d_results);
    thrust::device_ptr<double> d_results_end(d_results + template_mutations * numReads);

    // Create storage for P(all reads | mutated template) on GPU
    double *d_mutation_scores;
    cudaSafeCall( hipMalloc(&d_mutation_scores, template_mutations * sizeof(double)) );
    thrust::device_ptr<double> d_mutation_scores_start(d_mutation_scores);
    thrust::device_ptr<double> d_mutation_scores_end(d_mutation_scores + template_mutations);
    typedef thrust::device_ptr<double> thrust_device_double_ptr;
    strided_range<thrust_device_double_ptr> each_mutation(d_results_start + numReads - 1, d_results_end, numReads);

    // Create temp placeholder space for currentTmplSeq
    uint8_t *copyOfCurrentTmplSeq = (uint8_t*) malloc(tmplLen * sizeof(uint8_t));
    memset(copyOfCurrentTmplSeq, '\0', tmplLen * sizeof(uint8_t));

    /** Score Mutations **/

    // Figure out launch params
    dim3 score_threadsPerBlock(32, 8);

    int score_blocks_needed = (template_mutations * numReads + SCORE_THREADS_PER_BLOCK - 1) / SCORE_THREADS_PER_BLOCK;
    int score_block_x = (score_blocks_needed >= MAX_BLOCKS_PER_GRID ? MAX_BLOCKS_PER_GRID : score_blocks_needed);
    int score_block_y = (score_blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;

    dim3 score_blocksPerGrid(score_block_x, score_block_y);

    // Run it

    int k = 0;

    while (true) {

        cudaSafeCall( hipMemcpy(d_currentTmplSeq, polishedTmplSeq, tmplLen * sizeof(uint8_t), hipMemcpyHostToDevice) );
        cudaSafeCall( hipMemset(d_results, '\0', template_mutations * numReads * sizeof(double)) );
        cudaSafeCall( hipMemset(d_mutation_scores, '\0', template_mutations * sizeof(double)) );

        ScoreMutations<<<score_blocksPerGrid, score_threadsPerBlock>>>(d_currentTmplSeq, d_readSeqs, rs_pitch, d_qvInfo, tmplLen, readLen, numReads, d_results);

        // Add read log-probs for each mutation to get total prob for that mutation
        thrust::inclusive_scan(d_results_start, d_results_end, d_results_start);
        thrust::adjacent_difference(each_mutation.begin(), each_mutation.end(), d_mutation_scores_start);

        cudaSafeCall( hipMemcpy(results, d_mutation_scores, template_mutations * sizeof(double), hipMemcpyDeviceToHost) );
        //results[0] = k;
        // Find max element
        thrust::device_ptr<double> d_max_prob_mutation_pos = thrust::max_element(d_mutation_scores_start, d_mutation_scores_end);

        // If there's a higher scoring mutation, prepare to run again

        if (*d_max_prob_mutation_pos > currentTmplScore) {

            int mut_id = (d_max_prob_mutation_pos - d_mutation_scores_start);

            // Backup current template
            memcpy(copyOfCurrentTmplSeq, polishedTmplSeq, tmplLen * sizeof(uint8_t));
            indexToMutation(copyOfCurrentTmplSeq, tmplLen, polishedTmplSeq, mut_id);
            currentTmplScore = *d_max_prob_mutation_pos;

        } else {

            break;

        }
        k++;
    }

    free(copyOfCurrentTmplSeq);
    cudaSafeCall( hipFree(d_mutation_scores) );
    cudaSafeCall( hipFree(d_results) );
    cudaSafeCall( hipFree(d_qvInfo.ptr) );
    cudaSafeCall( hipFree(d_readSeqs) );
    cudaSafeCall( hipFree(d_currentTmplSeq) );

    return 0;
}
